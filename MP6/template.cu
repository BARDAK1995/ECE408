#include "hip/hip_runtime.h"
// MP Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ...
// +
// lst[n-1]}

#include <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

  __global__ void accumulateSums(float *DataArray, float *Sums, int len) {
    int index = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    int index2 = 2 * blockIdx.x * blockDim.x + threadIdx.x + blockDim.x;
    if((blockIdx.x > 0) && (index < len)) DataArray[index] += Sums[blockIdx.x-1];
    if((blockIdx.x > 0) && (index2 < len)) DataArray[index2] += Sums[blockIdx.x-1];
  }

__global__ void scan(float *input, float *output, float *AuxilarySum, int len) {
  const int sectionsize = blockDim.x*2;
  __shared__ float XY[BLOCK_SIZE*2];
  int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;
  // Load data into shared memory
  if(i < len) 
    XY[threadIdx.x] = input[i];
  else 
    XY[threadIdx.x] = 0.0f;
  if(i + blockDim.x < len) 
    XY[threadIdx.x + blockDim.x] = input[i + blockDim.x];
  else 
    XY[threadIdx.x + blockDim.x] = 0.0f;
  // Reduction forward
  for(int stride = 1; stride <= blockDim.x; stride *= 2) {
      __syncthreads();
      int index = (threadIdx.x + 1) * 2 * stride - 1;
      if(index < sectionsize && (index-stride) >= 0) { 
          XY[index] += XY[index - stride];
      }
  }
  // Traverse back up
  for (int stride = sectionsize / 4; stride > 0; stride /= 2) {
      __syncthreads();
      int index = (threadIdx.x + 1) * stride * 2 - 1;
      if(index + stride < sectionsize) {
          XY[index + stride] += XY[index];
      }
  }
  __syncthreads();
  // Write results back to global memory
  //if its the last thread of the block, make it do this work, we do this first to make the kernel usable for SUM scanning as well
  if(threadIdx.x == (blockDim.x-1)) AuxilarySum[blockIdx.x] =  XY[sectionsize-1];
  if (i < len) output[i] = XY[threadIdx.x];
  if ((i + blockDim.x) < len) output[i + blockDim.x] = XY[threadIdx.x + blockDim.x];
}


int main(int argc, char **argv) {
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  float *deviceSum;
  int numElements; // number of elements in the list
  int numBlocks;
  args = wbArg_read(argc, argv);
  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &numElements);
  numBlocks = (numElements - 1) / (BLOCK_SIZE * 2) + 1;                     // number of blocks
  hostOutput = (float *)malloc(numElements * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numElements);
  wbTime_start(GPU, "Allocating GPU memory.");
  wbCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceSum, numBlocks * sizeof(float)));       //allocating device sum memory
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Clearing output memory.");
  wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
  wbCheck(hipMemset(deviceSum, 0, numBlocks * sizeof(float)));
  wbTime_stop(GPU, "Clearing output memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float),
                     hipMemcpyHostToDevice));
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 dimBlock(BLOCK_SIZE, 1, 1);
  dim3 dimGrid(numBlocks, 1, 1); // Ensuring all elements are covered

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce
  scan<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, deviceSum, numElements);
  scan<<<dim3(1, 1, 1), dimBlock>>>(deviceSum, deviceSum, deviceSum, numBlocks);
  accumulateSums<<<dimGrid, dimBlock>>>(deviceOutput, deviceSum, numElements);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float),
                     hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput);
  hipFree(deviceSum);
  hipFree(deviceOutput);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, numElements);

  free(hostInput);
  free(hostOutput);

  return 0;
}

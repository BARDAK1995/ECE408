#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 16

void matrixMultiplyCPU(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns){
    for (int i = 0; i < numAColumns; i++)
    {
        for (int j = 0; j < numBRows; j++)
        {   
            float sum = 0;
            for (int k = 0; k < numCColumns; k++)
            { 
                float a = A[(i * numAColumns) + k];
                float b = B[j + (k * numBColumns)];
                sum += a*b;
            }
            C[(i * numCColumns) + j] = sum;
        }
    }
}


__global__ void matrixMultiplyGPU(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //   index values for the global matrix
  int column = blockDim.x * blockIdx.x + threadIdx.x;
  int row = blockDim.y * blockIdx.y + threadIdx.y;
  if (column < numCColumns && row < numCRows)
  {
    float cvalue = 0;
    for (int i = 0; i < numAColumns; i++)
    {
        const float a = A[(row * numAColumns) + i];
        const float b = B[column + (numBColumns * i)];
        cvalue += a * b;
    }
    C[column + (row * numCColumns)] = cvalue;
  }
}

__global__ void matrixMultiplyGPUTiled(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
  __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];
  const int width = numAColumns;
  const int Adepth = numARows;
  const int Bdepth = numBColumns;
  
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int blockx = blockDim.x;
  const int blocky = blockDim.y;
  //   index values for the global output matrix
  const int column = blockx * blockIdx.x + threadIdx.x;
  const int row = blocky * blockIdx.y + threadIdx.y;
  
  
  float Cvalue = 0.0f;
  const int n_tiles = ceil(width/(float)TILE_WIDTH);
  for (int tileNo = 0; tileNo < n_tiles; tileNo++){
    // load tile to shared memory
    // forA
    if ((row < Adepth) && ((tx + tileNo * TILE_WIDTH) < width))
        tileA[ty][tx] = A[(row * width) + tx + (tileNo * TILE_WIDTH)];
    else tileA[ty][tx] = 0.0f;
    // For B
    if ((column < Bdepth) && ((ty + tileNo * TILE_WIDTH) < width))
        tileB[ty][tx] = B[column + (ty * Bdepth) + (tileNo * Bdepth * TILE_WIDTH)];
    else tileB[ty][tx] = 0.0f;
    __syncthreads();
    // calculate partial multiplication result for this tile 
    for (int k = 0; k < TILE_WIDTH; k++)
        Cvalue += tileA[ty][k] * tileB[k][tx];
    __syncthreads();
  }
  //   put the correct summed up multiplication result
  if ((row < numARows) && (column < numBColumns))
    C[row*numCColumns + column] = Cvalue;
}

void writeMatrixToFile(const float* matrix, int numRows, int numColumns, const std::string& filename) {
    std::ofstream outfile(filename);
    if (!outfile.is_open()) {
        std::cerr << "Error: Could not open the file" << std::endl;
        return;
    }
    outfile << numRows << " " << numColumns << "\n";
    for (int i = 0; i < numRows; ++i) {
        for (int j = 0; j < numColumns; ++j) {
            outfile << matrix[i * numColumns + j];
            if (j < numColumns - 1) {
                outfile << "  ";
            }
        }
        outfile << '\n';
    }
    outfile.close();
}

float* readMatrixFromFile(const std::string& filePath, int* rows, int* cols) {
    std::ifstream file(filePath);
    if(!file) {
        std::cerr << "Could not open the file!" << std::endl;
        return nullptr;
    }
    file >> *rows >> *cols;

    if(!file || *rows <= 0 || *cols <= 0) {
        std::cerr << "Error reading matrix dimensions!" << std::endl;
        return nullptr;
    }
    float* matrix = (float*) malloc( (*rows) * (*cols) * sizeof(float));
    for(int i = 0; i < *rows; ++i) {
        for(int j = 0; j < *cols; ++j) {
            file >> matrix[i * *cols + j];
        }
    }
    if(file.fail()) {
        std::cerr << "Error reading file!" << std::endl;
        delete[] matrix;
        return nullptr;
    }
    file.close();
    return matrix;
}


int main() {
    const std::string filePathA = "./data/9/input0.raw";
    const std::string filePathB = "./data/9/input1.raw";

    float *hostA; // The A matrix
    float *hostB; // The B matrix
    float *hostC; // The output C matrix
    float *deviceA;
    float *deviceB;
    float *deviceC;
    int numARows;    // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A
    int numBRows;    // number of rows in the matrix B
    int numBColumns; // number of columns in the matrix B
    int numCRows;    // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set
                    // this)

    hostA = readMatrixFromFile(filePathA, &numARows, &numAColumns);
    hostB = readMatrixFromFile(filePathB, &numBRows, &numBColumns);
    //@@ Set numCRows and numCColumns
    numCRows = numARows;
    numCColumns = numBColumns;

    int sizeA = numARows * numAColumns;
    int sizeB = numBRows * numBColumns;
    int sizeC = numCRows * numCColumns;

    //@@ Allocate the hostC matrix
    hostC = (float*)malloc(sizeC * sizeof(float));
    // solutionCPU
    // matrixMultiplyCPU(hostA, hostB, hostC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
    // writeMatrixToFile(hostC, numCRows, numCColumns, "output.raw");
    
    //@@ Allocate GPU memory here
    hipMalloc((void**)&deviceA, sizeA * sizeof(float));
    hipMalloc((void**)&deviceB, sizeB * sizeof(float));
    hipMalloc((void**)&deviceC, sizeC * sizeof(float));
    //@@ Copy memory to the GPU here
    hipMemcpy(deviceA, hostA, sizeA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, sizeB * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceC, hostC, sizeC * sizeof(float), hipMemcpyHostToDevice);

    //@@ Initialize the grid and block dimensions here
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    const int gridXdim = ceil(numCColumns/(float)TILE_WIDTH);
    const int gridYdim = ceil(numCRows/(float)TILE_WIDTH);
    std::cout << gridXdim << gridYdim <<std::endl;
    dim3 DimGrid(gridXdim, gridYdim, 1);

    //@@ Launch the GPU Kernel here
    // matrixMultiplyGPU<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
    matrixMultiplyGPUTiled<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
    //@@ Copy the GPU memory back to the CPU here
    hipError_t errA = hipMemcpy(hostA, deviceA, sizeA * sizeof(float), hipMemcpyDeviceToHost);
    hipError_t errB = hipMemcpy(hostB, deviceB, sizeB * sizeof(float), hipMemcpyDeviceToHost);
    hipError_t errC = hipMemcpy(hostC, deviceC, sizeC * sizeof(float), hipMemcpyDeviceToHost);
    // if (errA != cudaSuccess || errB != cudaSuccess || errC != cudaSuccess) {
    //     fprintf("Failed to Allocate deviceA memory:");
    // }
    //@@ Free the GPU memory here
    writeMatrixToFile(hostC, numCRows, numCColumns, "output2.raw");

    free(hostA); free(hostB); free(hostC);
    hipFree(deviceA); hipFree(deviceB); hipFree(deviceC);

    return 0;
}

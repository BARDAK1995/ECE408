#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
__constant__ float KERNEL_DEVICE_CST[3136];

__global__ void conv_forward_kernel_basic(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]    // out_4d(b, m, h, w)
    #define in_4d_global(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]                          // in_4d(b, c, cell_height, cell_width)
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]                         // mask_4d(m, c, mask_heightindex, mask_widthindex)
    // Insert your GPU convolution kernel code here
    const int tile_width = blockDim.x;
    const int tile_height = blockDim.y;
    const int W_grid_blocks = (W_out - 1) / tile_width + 1;  //tiles in outputWidth
    const int m_feature = blockIdx.x;
    const int b = blockIdx.z;
    const int output_h = (blockIdx.y / W_grid_blocks) * tile_height + threadIdx.y;
    const int output_w = (blockIdx.y % W_grid_blocks) * tile_width + threadIdx.x;
    // starting index for current Block
    const int input_h_start = output_h * S; 
    const int input_w_start = output_w * S;
    int input_x;// input-x index
    int input_y;// input-y index
    float acc = 0.0f;
    if((output_h < H_out) && (output_w < W_out)){
        for(int c = 0; c < C; ++c){   // sum over all input channels
            for(int j = 0; j < K; ++j){   // KxK filter (height)
                input_y = input_h_start + j;
                for(int i = 0; i < K; ++i){   // KxK filter (width)
                    input_x = input_w_start + i;
                    acc += in_4d_global(b, c, input_y, input_x) * mask_4d(m_feature, c, j, i); 
                }
            }
        }
        out_4d(b, m_feature, output_h, output_w) = acc;
    }
    #undef out_4d
    #undef in_4d_global
    #undef mask_4d
}


__global__ void conv_forward_kernel_ConstantMem(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Function paramter definitions:
    output - output
    input - input
    KERNEL_DEVICE_CST - convolution kernel mask in constant MEM
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]    // out_4d(b, m, h, w)
    #define in_4d_global(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]                          // in_4d(b, c, cell_height, cell_width)
    #define mask_4d(i3, i2, i1, i0) KERNEL_DEVICE_CST[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]                         // mask_4d(m, c, mask_heightindex, mask_widthindex)
    // Insert your GPU convolution kernel code here
    const int tile_width = blockDim.x;
    const int tile_height = blockDim.y;

    const int W_grid_blocks = (W_out - 1) / tile_width + 1;  //tiles in outputWidth
    const int m_feature = blockIdx.x;
    const int b = blockIdx.z;
    const int output_h = (blockIdx.y / W_grid_blocks) * tile_height + threadIdx.y;
    const int output_w = (blockIdx.y % W_grid_blocks) * tile_width + threadIdx.x;
    // starting index for current Block
    const int input_h_start = output_h * S; 
    const int input_w_start = output_w * S;
    int input_x;// input-x index
    int input_y;// input-y index
    float acc = 0.0f;
    if((output_h < H_out) && (output_w < W_out)){
        for(int c = 0; c < C; ++c){   // sum over all input channels
            for(int j = 0; j < K; ++j){   // KxK filter (height)
                input_y = input_h_start + j;
                for(int i = 0; i < K; ++i){   // KxK filter (width)
                    input_x = input_w_start + i;
                    acc += in_4d_global(b, c, input_y, input_x) * mask_4d(m_feature, c, j, i); 
                }
            }
        }
        out_4d(b, m_feature, output_h, output_w) = acc;
    }
    #undef out_4d
    #undef in_4d_global
    #undef mask_4d
}

__global__ void conv_forward_kernel_ConstantMem_bigstride(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Function paramter definitions:
    output - output
    input - input
    KERNEL_DEVICE_CST - convolution kernel mask in constant MEM
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]    // out_4d(b, m, h, w)
    #define in_4d_global(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]                          // in_4d(b, c, cell_height, cell_width)
    #define mask_4d(i3, i2, i1, i0) KERNEL_DEVICE_CST[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]                         // mask_4d(m, c, mask_heightindex, mask_widthindex)
    // Insert your GPU convolution kernel code here
    const int tile_width = blockDim.x;
    const int tile_height = blockDim.y;
    const int W_grid_blocks = (W_out - 1) / tile_width + 1;  //tiles in outputWidth
    const int m_feature = blockIdx.x;
    const int b = blockIdx.z;
    const int output_h = (blockIdx.y / W_grid_blocks) * tile_height + threadIdx.y;
    const int output_w = (blockIdx.y % W_grid_blocks) * tile_width + threadIdx.x;
    // starting index for current Block
    const int input_h_start = output_h * S; 
    const int input_w_start = output_w * S;
    int input_x;// input-x index
    int input_y;// input-y index
    float acc = 0.0f;
    if((output_h < H_out) && (output_w < W_out)){
        for(int c = 0; c < C; ++c){   // sum over all input channels
            for(int j = 0; j < K; ++j){   // KxK filter (height)
                input_y = input_h_start + j;
                for(int i = 0; i < K; ++i){   // KxK filter (width)
                    input_x = input_w_start + i;
                    acc += in_4d_global(b, c, input_y, input_x) * mask_4d(m_feature, c, j, i); 
                }
            }
        }
        out_4d(b, m_feature, output_h, output_w) = acc;
    }
    #undef out_4d
    #undef in_4d_global
    #undef mask_4d
}
__global__ void conv_forward_kernel_ConstantMem_SharedMem(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Function paramter definitions:
    output - output
    input - input
    KERNEL_DEVICE_CST - convolution kernel mask in constant MEM
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */
    extern __shared__ float N_ds[]; //size determined dynamicly at runtime, we will rely on cache to catch others
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    const int tile_width = blockDim.x;
    const int tile_height = blockDim.y;
    const int SharedMatrix_width = tile_width * S;
    const int SharedMatrix_height = tile_height * S;
    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]    // out_4d(b, m, h, w)
    #define in_4d_global(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]                          // in_4d_global(b, c, cell_height, cell_width)
    #define in_4d_shared(i2, i1, i0) N_ds[(i2) * (SharedMatrix_height * SharedMatrix_width) + (i1) * (SharedMatrix_width) + i0]                          // in_4d_shared(c, cell_height, cell_width)
    #define mask_4d(i3, i2, i1, i0) KERNEL_DEVICE_CST[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]                         // mask_4d(m, c, mask_heightindex, mask_widthindex)
    // Insert your GPU convolution kernel code here
    const int W_grid_blocks = (W_out - 1) / tile_width + 1;  //tiles in outputWidth
    const int m_feature = blockIdx.x;
    const int b = blockIdx.z;
    // y and x indexes for the output matrix
    const int output_h = (blockIdx.y / W_grid_blocks) * tile_height + threadIdx.y;
    const int output_w = (blockIdx.y % W_grid_blocks) * tile_width + threadIdx.x;
    // corresponding y and x starting indexes for input matrix for the current block 
    const int input_y_Block_start = ((blockIdx.y / W_grid_blocks) * tile_height) * S; 
    const int input_x_Block_start = ((blockIdx.y % W_grid_blocks) * tile_width) * S;
    int input_x;// input-x index
    int input_y;// input-y index
    int shared_x;// shared-x index
    int shared_y;// shared-y index
    // starting index for current Block
    //load Shared Memory
    for(int c = 0; c < C; ++c){
        for (int scounterx = 0; scounterx < S; ++scounterx){
            shared_x = threadIdx.x + scounterx * tile_width;
            input_x = input_x_Block_start + shared_x;
            for (int scountery = 0; scountery < S; ++scountery){
                shared_y = threadIdx.y + scountery * tile_height;
                input_y = input_y_Block_start + shared_y;
                //INDEXING OVER C in the outermost layer, to not mess up the coalescedd memory acces
                if((input_y > H) && (input_x > W)){
                    in_4d_shared(c, shared_y, shared_x) = 0.0f;
                }
                else {
                    in_4d_shared(c, shared_y, shared_x) = in_4d_global(b, c, input_y, input_x);
                }  
            }
        }
    }
    __syncthreads();
    const int input_h_start = output_h * S; 
    const int input_w_start = output_w * S;
    float acc = 0.0f;
    if((output_h < H_out) && (output_w < W_out)){
        for(int c = 0; c < C; ++c){   // sum over all input channels
            for(int j = 0; j < K; ++j){   // KxK filter (height)
                input_y = input_h_start + j;
                shared_y = input_y - input_y_Block_start; //where it is in corresponding input tile, we use this to determine if its in shared mem or not.
                for(int i = 0; i < K; ++i){   // KxK filter (width)
                    input_x = input_w_start + i;
                    shared_x = input_x - input_x_Block_start; //where it is in corresponding input tile, we use this to determine if its in shared mem or not.
                    if((shared_y<SharedMatrix_height) && (shared_x<SharedMatrix_width)){
                        acc += in_4d_shared(c, shared_y, shared_x) * mask_4d(m_feature, c, j, i); 
                    }
                    else{
                        acc += in_4d_global(b, c, input_y, input_x) * mask_4d(m_feature, c, j, i); 
                    }
                }
            }
        }
        out_4d(b, m_feature, output_h, output_w) = acc;
    }
    #undef out_4d
    #undef in_4d_global
    #undef mask_4d
}

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU
    const int memSizeInput = (B * C * H * W) * sizeof(float);
    const int memSizeMask = (M * C * K * K) * sizeof(float);
    // std::cout << "The value of memSizeMask is: " << memSizeMask << std::endl;
    const int outputHeight = (H - K)/S + 1;
    const int outputWidth = (W - K)/S + 1;
    const int memSizeOutput = (B * M * outputHeight * outputWidth) * sizeof(float);
    // std::cout << "channel size: " << C << "kernel width is: " << K << "stride  is: " << S <<std::endl;
    // std::cout << "output height is:  " << outputHeight << "output width is: " << outputWidth << "Channel is: " << C << "stride  is: " << S << std::endl;

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.
    hipMalloc((void **)device_input_ptr, memSizeInput);
    hipMalloc((void **)device_mask_ptr, memSizeMask);
    hipMalloc((void **)device_output_ptr, memSizeOutput);
    hipMemcpy(*device_input_ptr, host_input, memSizeInput, hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, memSizeMask, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(KERNEL_DEVICE_CST), host_mask, memSizeMask);


    // get_device_properties();
    // // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    const int memSizeMask = (M * C * K * K) * sizeof(float);
    const int outputHeight = (H - K)/S + 1;
    const int outputWidth = (W - K)/S + 1;
    int TILE_WIDTH = 6;
    int TILE_HEIGHT = 48;
    int H_grid_blocks = (outputHeight - 1) / TILE_HEIGHT + 1; //tiles in outputHeight
    int W_grid_blocks = (outputWidth - 1) / TILE_WIDTH + 1;  //tiles in outputWidth
    int nTiles = H_grid_blocks * W_grid_blocks; // total tiles
    int sharedMemConvSize = (TILE_WIDTH * TILE_HEIGHT * S * S * C) * sizeof(float);
    while (sharedMemConvSize > 49152){
        TILE_HEIGHT /= 2;
        // W_grid_blocks = (outputWidth - 1) / TILE_WIDTH + 1;
        H_grid_blocks = (outputHeight - 1) / TILE_HEIGHT + 1; //tiles in outputHeight
        nTiles = H_grid_blocks * W_grid_blocks; // total tiles
        sharedMemConvSize = (TILE_WIDTH * TILE_HEIGHT * S * S * C) * sizeof(float);
        std::cout<<"REsizing _________"<<std::endl;
    }
    dim3 dimBlock(TILE_WIDTH, TILE_HEIGHT, 1);
    dim3 dimGrid(M, nTiles, B); // Ensuring all elements are covered
    // std::cout << "The memsize of sharedELementMatrix is: " << sharedMemConvSize << std::endl;     // max size is 49152

    if(S>=K){
        // conv_forward_kernel_ConstantMem_bigstride<<<dimGrid, dimBlock>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
        conv_forward_kernel_ConstantMem_SharedMem<<<dimGrid, dimBlock, sharedMemConvSize>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
        // std::cout<<"BIGGSTRIDE"<<std::endl;
    }
    else{
        // conv_forward_kernel_basic<<<dimGrid, dimBlock>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
        // conv_forward_kernel_ConstantMem<<<dimGrid, dimBlock>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
        conv_forward_kernel_ConstantMem_SharedMem<<<dimGrid, dimBlock, sharedMemConvSize>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
    }

}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Copy the output back to host
    
    const int outputHeight = (H - K)/S + 1;
    const int outputWidth = (W - K)/S + 1;
    const int memSizeOutput = (B * M * outputHeight * outputWidth) * sizeof(float);
    hipMemcpy(host_output, device_output, memSizeOutput,hipMemcpyDeviceToHost);
    hipFree(device_input);
    hipFree(device_mask);
    hipFree(device_output);
    // Free device memory
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}

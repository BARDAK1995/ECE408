#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#include <hip/hip_fp16.h>
// #include <chrono>
hipStream_t stream1;
__constant__ half KERNEL_DEVICE_CST[3136];

// __global__ void conv_forward_kernel_basic(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
// {
//     /*
//     Function paramter definitions:
//     output - output
//     input - input
//     mask - convolution kernel
//     B - batch_size (number of images in x)
//     M - number of output feature maps
//     C - number of input feature maps
//     H - input height dimension
//     W - input width dimension
//     K - kernel height and width (K x K)
//     S - stride step length
//     */
//     const int H_out = (H - K)/S + 1;
//     const int W_out = (W - K)/S + 1;
//     #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]    // out_4d(b, m, h, w)
//     #define in_4d_global(i3, i2, i1, i0) (input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0])     // in_4d(b, c, cell_height, cell_width)
//     #define mask_4d(i3, i2, i1, i0) (KERNEL_DEVICE_CST[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0])                         // mask_4d(m, c, mask_heightindex, mask_widthindex)
//     // Insert your GPU convolution kernel code here
//     const int tile_width = blockDim.x;
//     const int tile_height = blockDim.y;
//     const int W_grid_blocks = (W_out - 1) / tile_width + 1;  //tiles in outputWidth
//     const int m_feature = blockIdx.x;
//     const int b = blockIdx.z;
//     const int output_h = (blockIdx.y / W_grid_blocks) * tile_height + threadIdx.y;
//     const int output_w = (blockIdx.y % W_grid_blocks) * tile_width + threadIdx.x;
//     // starting index for current Block
//     const int input_h_start = output_h * S; 
//     const int input_w_start = output_w * S;
//     int input_x;// input-x index
//     int input_y;// input-y index
//     float acc = 0.0f;
//     if((output_h < H_out) && (output_w < W_out)){
//         for(int c = 0; c < C; ++c){   // sum over all input channels
//             for(int j = 0; j < K; ++j){   // KxK filter (height)
//                 input_y = input_h_start + j;
//                 for(int i = 0; i < K; ++i){   // KxK filter (width)
//                     input_x = input_w_start + i;
//                     acc += in_4d_global(b, c, input_y, input_x) * mask_4d(m_feature, c, j, i); 
//                 }
//             }
//         }
//         out_4d(b, m_feature, output_h, output_w) = acc;
//     }
//     #undef out_4d
//     #undef in_4d_global
//     #undef mask_4d
// }

__global__ void conv_forward_kernel_basic_16FP(float* __restrict__ output, const half* __restrict__ input, const half* __restrict__ mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]    // out_4d(b, m, h, w)
    #define in_4d_global(i3, i2, i1, i0) (input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0])     // in_4d(b, c, cell_height, cell_width)
    #define mask_4d(i3, i2, i1, i0) (KERNEL_DEVICE_CST[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0])                         // mask_4d(m, c, mask_heightindex, mask_widthindex)
    // Insert your GPU convolution kernel code here
    const int tile_width = blockDim.x;
    const int tile_height = blockDim.y;
    const int W_grid_blocks = (W_out - 1) / tile_width + 1;  //tiles in outputWidth
    const int m_feature = blockIdx.x;
    const int b = blockIdx.z;
    const int output_h = (blockIdx.y / W_grid_blocks) * tile_height + threadIdx.y;
    const int output_w = (blockIdx.y % W_grid_blocks) * tile_width + threadIdx.x;
    // starting index for current Block
    const int input_h_start = output_h * S; 
    const int input_w_start = output_w * S;
    int input_x;// input-x index
    int input_y;// input-y index
    half acc = __float2half(0.0f);
    // float acc = 0.0f;
    if((output_h < H_out) && (output_w < W_out)){
        for(int c = 0; c < C; ++c){   // sum over all input channels
            for(int j = 0; j < K; ++j){   // KxK filter (height)
                input_y = input_h_start + j;
                for(int i = 0; i < K; ++i){   // KxK filter (width)
                    input_x = input_w_start + i;
                    acc = __hadd(acc, __hmul(in_4d_global(b, c, input_y, input_x), mask_4d(m_feature, c, j, i)));
                    // acc += in_4d_global(b, c, input_y, input_x) * mask_4d(m_feature, c, j, i);
                    // acc = __hfma(in_4d_global(b, c, input_y, input_x), mask_4d(m_feature, c, j, i), acc);
                }
            }
        }
        out_4d(b, m_feature, output_h, output_w) = __half2float(acc);
    }
    #undef out_4d
    #undef in_4d_global
    #undef mask_4d
}

__global__ void conv_forward_kernel_basic_16FP_convLayerK7_CnstMask(float* __restrict__ output, const half* __restrict__ input, const half* __restrict__ mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]    // out_4d(b, m, h, w)
    #define in_4d_global(i3, i2, i1, i0) (input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0])     // in_4d(b, c, cell_height, cell_width)
    #define mask_4d(i3, i2, i1, i0) (KERNEL_DEVICE_CST[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0])                         // mask_4d(m, c, mask_heightindex, mask_widthindex)
    // Insert your GPU convolution kernel code here
    const int tile_width = blockDim.x;
    const int tile_height = blockDim.y;
    const int W_grid_blocks = (W_out - 1) / tile_width + 1;  //tiles in outputWidth
    const int m_feature = blockIdx.x;
    const int b = blockIdx.z;
    const int output_h = (blockIdx.y / W_grid_blocks) * tile_height + threadIdx.y;
    const int output_w = (blockIdx.y % W_grid_blocks) * tile_width + threadIdx.x;
    // starting index for current Block
    const int input_h_start = output_h * S; 
    const int input_w_start = output_w * S;
    int input_x;// input-x index
    int input_y;// input-y index
    half acc = __float2half(0.0f);
    if((output_h < H_out) && (output_w < W_out)){
        for(int c = 0; c < C; ++c){   // sum over all input channels
            #pragma unroll 7
            for(int j = 0; j < K; ++j){   // KxK filter (height)
                input_y = input_h_start + j;
                #pragma unroll 7
                for(int i = 0; i < K; ++i){   // KxK filter (width)
                    input_x = input_w_start + i;
                    acc = __hadd(acc, __hmul(in_4d_global(b, c, input_y, input_x), mask_4d(m_feature, c, j, i)));
                    // acc += in_4d_global(b, c, input_y, input_x) * mask_4d(m_feature, c, j, i);
                }
            }
        }
        out_4d(b, m_feature, output_h, output_w) = __half2float(acc);
    }
    #undef out_4d
    #undef in_4d_global
    #undef mask_4d
}
// converts arrays to half in gpu
__global__ void convertFloatToHalf(half *output, const float *input, const int numElements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numElements) {
        output[idx] = __float2half(input[idx]);
    }
}

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    hipStreamCreate(&stream1);
    // Allocate memory and copy over the relevant data structures to the GPU
    const int nInputElements = (B * C * H * W);
    const int memSizeInput = nInputElements * sizeof(float);
    const int mMaskElements = (M * C * K * K);
    const int memSizeMask = mMaskElements * sizeof(float);
    // std::cout << mMaskElements << "   n mask elements " << std::endl;
    const int outputHeight = (H - K)/S + 1;
    const int outputWidth = (W - K)/S + 1;
    const int nOutputElements = (B * M * outputHeight * outputWidth);
    const int memSizeOutput = nOutputElements * sizeof(float);
    
    // hipStreamCreate(&stream2);
    hipMalloc((void **)device_input_ptr, memSizeInput);
    hipMalloc((void **)device_mask_ptr, memSizeMask);
    // hipMemcpyToSymbol(HIP_SYMBOL(KERNEL_DEVICE_CST), host_mask, memSizeMask);
    hipMemcpyAsync(*device_input_ptr, host_input, memSizeInput, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(*device_mask_ptr, host_mask, memSizeMask, hipMemcpyHostToDevice, stream1);
    hipHostRegister(const_cast<float*>(host_output), memSizeOutput, hipHostRegisterDefault);
    hipMalloc((void **)device_output_ptr, memSizeOutput);
    // std::cout<<"mMaskElements: "<<mMaskElements<<std::endl;
    // auto start6 = std::chrono::high_resolution_clock::now();
    // hipHostRegister(const_cast<float*>(host_output), memSizeOutput, hipHostRegisterDefault);
    // auto stop6 = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<double> duration6 = stop6 - start6;
    // std::cout << "Output memory Pinning took " << duration6.count()*1000 << " ms" << std::endl;
    // get_device_properties();
    // // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    const int mMaskElements = (M * C * K * K);
    const int memSizeMask = mMaskElements * sizeof(float);
    const int memSizeMaskHalf = mMaskElements * sizeof(half);

    half *device_input_half;
    half *device_mask_half;
    const int nInputElements = (B * C * H * W);
    const int memSizeInput_half = nInputElements * sizeof(half);

    hipMalloc((void **)&device_input_half, memSizeInput_half);
    hipMalloc((void **)&device_mask_half, memSizeMaskHalf);
    const int blockSizeFP16Converter = 128;
    const int blockSizeFP16mask = 32;
    const int gridSizeFP16ConverterInput = (nInputElements + blockSizeFP16Converter - 1) / blockSizeFP16Converter;
    const int gridSizeFP16ConverterMask = (mMaskElements + blockSizeFP16Converter - 1) / blockSizeFP16mask;
    convertFloatToHalf<<<gridSizeFP16ConverterMask, blockSizeFP16mask, 0, stream1>>>(device_mask_half, device_mask, mMaskElements);
    convertFloatToHalf<<<gridSizeFP16ConverterInput, blockSizeFP16Converter, 0, stream1>>>(device_input_half, device_input, nInputElements);
    hipMemcpyToSymbol(HIP_SYMBOL(KERNEL_DEVICE_CST), device_mask_half, memSizeMaskHalf);

    
    const int outputHeight = (H - K)/S + 1;
    const int outputWidth = (W - K)/S + 1;
    // std::cout << outputWidth << " x " << outputHeight << " x " << C << " and K is " << K << " and S is " << S << std::endl;
    int TILE_WIDTH = 6;
    int TILE_HEIGHT = 48;
    if(outputWidth==80){
        TILE_WIDTH = 16;
        TILE_HEIGHT = 16;
    }
    else if(outputWidth==34){
        TILE_WIDTH = 8;
        TILE_HEIGHT = 48;
    }
    int H_grid_blocks = (outputHeight - 1) / TILE_HEIGHT + 1; //tiles in outputHeight
    int W_grid_blocks = (outputWidth - 1) / TILE_WIDTH + 1;  //tiles in outputWidth
    int nTiles = H_grid_blocks * W_grid_blocks; // total tiles
    // int sharedMemConvSize = (TILE_WIDTH * TILE_HEIGHT * S * S * C) * sizeof(float);
    // while (sharedMemConvSize > 49152){
    //     TILE_HEIGHT /= 2;
    //     // W_grid_blocks = (outputWidth - 1) / TILE_WIDTH + 1;
    //     H_grid_blocks = (outputHeight - 1) / TILE_HEIGHT + 1; //tiles in outputHeight
    //     nTiles = H_grid_blocks * W_grid_blocks; // total tiles
    //     sharedMemConvSize = (TILE_WIDTH * TILE_HEIGHT * S * S * C) * sizeof(float);
    //     // std::cout<<"REsizing "<<std::endl;
    // }
    dim3 dimBlock(TILE_WIDTH, TILE_HEIGHT, 1);
    dim3 dimGrid(M, nTiles, B); // Ensuring all elements are covered
    // hipStreamSynchronize(stream1);
    if(K==7){
        conv_forward_kernel_basic_16FP_convLayerK7_CnstMask<<<dimGrid, dimBlock, 0, stream1>>>(device_output, device_input_half, device_mask_half, B, M, C, H, W, K, S);
    }
    else{
        conv_forward_kernel_basic_16FP<<<dimGrid, dimBlock, 0 , stream1>>>(device_output, device_input_half, device_mask_half, B, M, C, H, W, K, S);
    }
}

__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Calculate output size and memory size for half-precision
    const int outputHeight = (H - K)/S + 1;
    const int outputWidth = (W - K)/S + 1;
    const int nOutputElements = (B * M * outputHeight * outputWidth);
    const int memSizeOutput = nOutputElements * sizeof(float);
    // hipHostRegister(host_output, memSizeOutput, hipHostRegisterDefault);
    // hipStreamSynchronize(stream1);
    hipMemcpyAsync(host_output, device_output, memSizeOutput, hipMemcpyDeviceToHost,stream1);
    hipHostUnregister(host_output);
    
    // auto start4 = std::chrono::high_resolution_clock::now();
    // hipMemcpy(host_output, device_output, memSizeOutput, hipMemcpyDeviceToHost);
    // hipHostUnregister(host_output);
    // auto stop4 = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<double> duration4 = stop4 - start4;
    // std::cout << "FinalMemOps took " << duration4.count()*1000 << " ms" << std::endl;
    
    // Free device memory
    hipFree(device_input);
    hipFree(device_mask);
    hipFree(device_output);
    hipStreamDestroy(stream1);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}

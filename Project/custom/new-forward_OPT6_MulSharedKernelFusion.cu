#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#include <hip/hip_fp16.h>
__constant__ half KERNEL_DEVICE_CST[3136];
#define TILE_WIDTH_MATMUL 64
__global__ void matrixMultiplySharedFusion_unroll(float* __restrict__ OUTPUT_C, half* __restrict__ inputX, const int B, const int M, const int C, const int H, const int W, const int K, const int S) {
    extern __shared__ half tileAB[];  // Declaration of the shared memory array
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    // matrix sizes
    const int B_width = H_out*W_out; //numBColumns;
    const int B_height = C * K * K; // C*K*K; //numBrows;
    const int TILE_depth = C * K * K; //TILE_depth;
    const int Aheight = M; // numARows;
    const int A_width = C * K * K;
    const int sharedMatmulA_Nelements = A_width*M;

    const int WIDTH_unroll_tile = TILE_WIDTH_MATMUL;
    const int HIGHT_unroll = C * K * K;
    #define in_4d_global(i3, i2, i1, i0) inputX[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]     // in_4d(b, c, cell_height, cell_width)
    #define A_2d(i1, i0) (KERNEL_DEVICE_CST[(i1) * (C*K*K) + i0]) // mask_4d(m, c, mask_heightindex, mask_widthindex) = mask_4d(m=y, x)
    #define B_2d_shared(i1, i0) tileAB[ (i1) * (TILE_WIDTH_MATMUL) + i0 + sharedMatmulA_Nelements]     // outputUnrolles(b, cell_height, cell_width)
    #define A_2d_shared(i1, i0) tileAB[ (i1) * (A_width) + i0]     // outputUnrolles(b, cell_height, cell_width)
    #define C_4d(i3, i2, i1) OUTPUT_C[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1)]    // out_4d(b, m, h, w)

    //   index values for the global output matrix
    const int column_X_outputElement = blockDim.x * blockIdx.x + threadIdx.x;
    const int row_Y_m_feature = blockDim.y * blockIdx.y + threadIdx.y;
    const int batchN = blockIdx.z;
    half Cvalue =  __float2half(0.0f);
    const int nTilesA = ceil(A_width / (float)(blockDim.x)); // load tile to shared memory For A
    for (int tileNoX = 0; tileNoX < nTilesA; tileNoX++){
        const int colxx = (tileNoX * blockDim.x) + threadIdx.x;
        if ((colxx < A_width) && (row_Y_m_feature < Aheight))
            A_2d_shared(threadIdx.y, colxx) = A_2d(row_Y_m_feature, colxx);
    }
    // __syncthreads();
    //Unroll B into shared
    const int Unroll_thread = blockIdx.x * blockDim.x + threadIdx.x;
    const int Unroll_channel = threadIdx.y;
    if ((Unroll_thread < B_width) && (Unroll_channel < C)) {
        // Channel of the input feature map being collected by the thread
        const int cc = Unroll_channel; 
        // const int x_unroll = thread % WIDTH_unroll;
        const int x_unroll = Unroll_thread;
        // Horizontal and vertical indices of the output elementss
        const int h_out = S * (x_unroll / W_out);
        const int w_out = S * (x_unroll % W_out);
        // Starting row index for the unrolled matrix section for channel c
        const int y_base_unrolled = cc * K * K;
        // #pragma unroll
        for(int q = 0; q < K; q++) {
            #pragma unroll 7
            for(int p = 0; p < K; p++) {
                const int input_y = h_out + p;
                const int y_unroll = y_base_unrolled + (p * K) + q;
                const int input_x = w_out + q;
                B_2d_shared(y_unroll, threadIdx.x) = in_4d_global(batchN, cc, input_y, input_x);
            }
        }
    }
    __syncthreads();
    // calculate partial multiplication result for this tile
    #pragma unroll 7
    for (int kk = 0; kk < TILE_depth; kk++){
        const half a = A_2d_shared(threadIdx.y, kk);
        const half b = B_2d_shared(kk, threadIdx.x);
        Cvalue = __hadd(Cvalue, __hmul(a, b));
    }
    __syncthreads();
    //   put the correct summed up multiplication result
    if ((row_Y_m_feature < Aheight) && (column_X_outputElement < B_width))
        C_4d(batchN, row_Y_m_feature, column_X_outputElement) = __half2float(Cvalue);
    #undef in_4d_global
    #undef A_2d
    #undef B_2d_shared
    #undef A_2d_shared
    #undef C_4d
}

// converts arrays to half in gpu
__global__ void convertFloatToHalf(half *output, const float *input, const int numElements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numElements) {
        output[idx] = __float2half(input[idx]);
    }
}

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU
    const int outputHeight = (H - K)/S + 1;
    const int outputWidth = (W - K)/S + 1;
    //input sizes
    const int nInputElements = (B * C * H * W);
    const int memSizeInput = nInputElements * sizeof(float);
    const int mMaskElements = (M * C * K * K);
    const int memSizeMask = mMaskElements * sizeof(float);
    // std::cout <<"Hin =" << H << "Win = " << W << "batch =" << B << "Chanelinput =" << C << std::endl;
    // std::cout <<"H out =" << outputHeight << "Wout = " << outputWidth << "output features =" << M << "Kernelsize =" << K << std::endl;
    //OutputSizes
    const int nOutputElements = (B * M * outputHeight * outputWidth);
    const int memSizeOutput = nOutputElements * sizeof(float);
    // std::cout << mMaskElements << "   n mask elements " << std::endl;
    hipMalloc((void **)device_input_ptr, memSizeInput);
    hipMalloc((void **)device_mask_ptr, memSizeMask);
    // hipMemcpyToSymbol(HIP_SYMBOL(KERNEL_DEVICE_CST), host_mask, memSizeMask);
    hipMemcpyAsync(*device_input_ptr, host_input, memSizeInput, hipMemcpyHostToDevice);
    hipMemcpyAsync(*device_mask_ptr, host_mask, memSizeMask, hipMemcpyHostToDevice);
    hipMalloc((void **)device_output_ptr, memSizeOutput);
    // std::cout<<"B is : "<<B<<" M is : "<<M<<std::endl;
    // std::cout << "Output memory Pinning took " << duration6.count()*1000 << " ms" << std::endl;
    
    // // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
}

__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    const int mMaskElements = (M * C * K * K);
    // const int memSizeMask = mMaskElements * sizeof(float);
    const int memSizeMaskHalf = mMaskElements * sizeof(half);

    half *device_input_half;
    half *device_mask_half;
    const int nInputElements = (B * C * H * W);
    const int memSizeInput_half = nInputElements * sizeof(half);

    hipMalloc((void **)&device_input_half, memSizeInput_half);
    hipMalloc((void **)&device_mask_half, memSizeMaskHalf);
    const int blockSizeFP16Converter = 128;
    const int blockSizeFP16mask = 32;
    const int gridSizeFP16ConverterInput = (nInputElements + blockSizeFP16Converter - 1) / blockSizeFP16Converter;
    const int gridSizeFP16ConverterMask = (mMaskElements + blockSizeFP16Converter - 1) / blockSizeFP16mask;
    convertFloatToHalf<<<gridSizeFP16ConverterMask, blockSizeFP16mask, 0, 0>>>(device_mask_half, device_mask, mMaskElements);
    hipMemcpyToSymbol(HIP_SYMBOL(KERNEL_DEVICE_CST), device_mask_half, memSizeMaskHalf);
    convertFloatToHalf<<<gridSizeFP16ConverterInput, blockSizeFP16Converter, 0, 0>>>(device_input_half, device_input, nInputElements);
    //Pointers to acces the fp16 portion of the arrays
    const int outputHeight = (H - K)/S + 1;
    const int outputWidth = (W - K)/S + 1;
    //Unrolled X matrix 
    // const int unrolledMatrixSize = B * C * K * K * outputHeight * outputWidth;
    // const int memSizeunrolledX = unrolledMatrixSize * sizeof(half);
    // std::cout <<"MATRIX SIZE is =" << outputHeight <<  std::endl;
    // half *device_UnrolledX;
    
    // hipMalloc((void **)&device_UnrolledX, memSizeunrolledX);
    // int blocksizeUnroll = 32;
    // int blocksPerInput = (outputHeight*outputWidth*C - 1) / blocksizeUnroll + 1; //tiles in outputHeight
    // dim3 dimGridUnroll(blocksPerInput, B, 1); // Ensuring all elements are covered
    // dim3 dimBlockUnroll(blocksizeUnroll, 1, 1);
    // hipStreamSynchronize(stream1);
    // unroll_Kernel<<<dimGridUnroll, dimBlockUnroll,0,0>>>(device_UnrolledX, device_input_half, B, C, H, W, K, S);
    // std::cout <<"Output MATRIX SIZE is =" << M << "x" << outputWidth*outputHeight << std::endl;
    // std::cout <<"CxKxK =" << C*K*K << std::endl;


    const int Matmul_Output_height = M;
    const int Matmul_Output_width = outputHeight * outputWidth;
    //for shared tiled matrix multiplyu
    int grid_blocks_X = (Matmul_Output_width - 1) / TILE_WIDTH_MATMUL + 1; // TILE_WIDTH_MATMUL = 32
    int grid_blocks_Y = (Matmul_Output_height - 1) / M + 1; //tiles in outputHeight should be 1
    dim3 DimBlock_sharedMatmul(TILE_WIDTH_MATMUL, M, 1); // TILE_WIDTH_MATMUL=32
    dim3 DimGrid_sharedMatmul(grid_blocks_X, grid_blocks_Y, B);
    const int sharedMatmulBsize = C*K*K*TILE_WIDTH_MATMUL * sizeof(half);
    const int sharedMatmulAsize = C*K*K*M * sizeof(half);
    const int sharedMem = sharedMatmulAsize + sharedMatmulBsize;
    //@@ Launch the GPU Kernel here
    matrixMultiplySharedFusion_unroll<<<DimGrid_sharedMatmul,DimBlock_sharedMatmul,sharedMem,0>>>(device_output, device_input_half, B, M, C, H, W, K, S);
    hipFree(device_input_half);
    hipFree(device_mask_half);
}

__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Calculate output size and memory size for half-precision
    const int outputHeight = (H - K)/S + 1;
    const int outputWidth = (W - K)/S + 1;
    const int nOutputElements = (B * M * outputHeight * outputWidth);
    const int memSizeOutput = nOutputElements * sizeof(float);
    hipHostRegister(host_output, memSizeOutput, hipHostRegisterDefault);
    // hipStreamSynchronize(stream1);
    hipMemcpyAsync(host_output, device_output, memSizeOutput, hipMemcpyDeviceToHost);
    // auto start4 = std::chrono::high_resolution_clock::now();
    // hipMemcpy(host_output, device_output, memSizeOutput, hipMemcpyDeviceToHost);
    // hipHostUnregister(host_output);
    // hipStreamDestroy(stream1);
    // Free device memory
    hipFree(device_input);
    hipFree(device_mask);
    hipFree(device_output);
    hipHostUnregister(host_output);

}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
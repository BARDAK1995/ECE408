#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define MASK_DIM 3
#define FilterRadius 1
#define input_TILE_WIDTH 8
#define O_TILE_WIDTH ((input_TILE_WIDTH) - 2 * (FilterRadius))
//@@ Define constant memory for device kernel here
__constant__ float filter3d_device[MASK_DIM][MASK_DIM][MASK_DIM];
__global__ void conv3d(float *input, float *output, const int z_size, const int y_size, const int x_size) {
  //@@ Insert kernel code here
  const int tx = threadIdx.x; 
  const int ty = threadIdx.y; 
  const int tz = threadIdx.z;
  const int col_i = blockIdx.x * O_TILE_WIDTH + tx - FilterRadius;
  const int row_i = blockIdx.y * O_TILE_WIDTH + ty - FilterRadius;
  const int depth_i = blockIdx.z * O_TILE_WIDTH + tz - FilterRadius;
  const int global_index = depth_i * x_size * y_size + row_i * x_size + col_i;
  // load shared mem
  __shared__ float N_ds[input_TILE_WIDTH][input_TILE_WIDTH][input_TILE_WIDTH];
  //load nonghost elements, put zero if ghost
  if ((row_i >= 0) && (row_i < y_size) && (col_i >= 0) && (col_i < x_size) && (depth_i >= 0) && (depth_i < z_size)) {
    N_ds[tx][ty][tz] = input[global_index];
  }
  else{
    N_ds[tx][ty][tz] = 0.0f;
  }
  __syncthreads();
  //  check if actual output cell is inside
  if ((row_i >= 0) && (row_i < y_size) && (col_i >= 0) && (col_i < x_size) && (depth_i >= 0) && (depth_i < z_size)){
    // then check if it really is a output point, and not a input memory loading thread
    if (( tx >= FilterRadius) && (tx < O_TILE_WIDTH + FilterRadius) &&
         (ty >= FilterRadius) && (ty < O_TILE_WIDTH + FilterRadius) && 
         (tz >= FilterRadius) && (tz < O_TILE_WIDTH + FilterRadius)){
      float Pvalue = 0.0f;
      for(int i=0; i < MASK_DIM; i++){
        for(int j=0; j < MASK_DIM; j++){
          for(int k=0; k < MASK_DIM; k++){
            Pvalue += filter3d_device[i][j][k] * N_ds[tx + i - FilterRadius][ty + j - FilterRadius][tz + k - FilterRadius];
          }
        }
      }
      output[global_index] = Pvalue;
    }
  }
  __syncthreads();
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  float *deviceInput;
  float *deviceOutput;
  args = wbArg_read(argc, argv);
  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));
  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);
  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");
  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ Allocate GPU memory here
  const int datalength = inputLength - 3;                                           // Recall that inputLength is 3 elements longer than the input data because the first  three elements were the dimensions
  hipMalloc((void**)&deviceInput, datalength*sizeof(float));
  hipMalloc((void**)&deviceOutput, datalength*sizeof(float));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  //@@ Copy input and kernel to GPU here
  hipMemcpyToSymbol(HIP_SYMBOL(filter3d_device), hostKernel, kernelLength*sizeof(float));
  hipMemcpy(deviceInput, hostInput+3, datalength*sizeof(float), hipMemcpyHostToDevice);   // Recall that the first three elements of hostInput are dimensions and
  hipMemcpy(deviceOutput, hostOutput+3, datalength*sizeof(float), hipMemcpyHostToDevice); // do not need to be copied to the gpu
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ Initialize grid and block dimensions here
  const int gridXdim = ceil(x_size / (float)O_TILE_WIDTH);
  const int gridYdim = ceil(y_size / (float)O_TILE_WIDTH);
  const int gridZdim = ceil(z_size / (float)O_TILE_WIDTH);
  dim3 dimGrid(gridXdim, gridYdim, gridZdim);
  dim3 dimBlock(input_TILE_WIDTH, input_TILE_WIDTH, input_TILE_WIDTH);
  //@@ Launch the GPU kernel here
  conv3d<<<dimGrid,  dimBlock>>>(deviceInput, deviceOutput, z_size, y_size, x_size);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ Copy the device memory back to the host here
  hipMemcpy(hostInput+3, deviceInput, datalength*sizeof(float), hipMemcpyDeviceToHost);            // Recall that the first three elements of the output are the dimensions
  hipMemcpy(hostOutput+3, deviceOutput, datalength*sizeof(float), hipMemcpyDeviceToHost);          // and should not be set here (they are set below)
  wbTime_stop(Copy, "Copying data from the GPU");
  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");
  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);
  // Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);
  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}
